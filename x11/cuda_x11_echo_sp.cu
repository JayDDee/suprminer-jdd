#include "hip/hip_runtime.h"
/*
	Based on Tanguy Pruvot's repo
	Provos Alexis - 2016
*/

#include "cuda_helper_alexis.h"
#include "cuda_vectors_alexis.h"

#define INTENSIVE_GMF
#include "cuda_x11_aes_sp.cuh"

__device__
static void echo_round_sp(const uint32_t sharedMemory[8 * 1024], uint32_t *W, uint32_t &k0){
	// Big Sub Words
#pragma unroll 16
	for (int idx = 0; idx < 16; idx++)
		AES_2ROUND_32(sharedMemory, W[(idx << 2) + 0], W[(idx << 2) + 1], W[(idx << 2) + 2], W[(idx << 2) + 3], k0);

	// Shift Rows
#pragma unroll 4
	for (int i = 0; i < 4; i++){
		uint32_t t[4];
		/// 1, 5, 9, 13
		t[0] = W[i + 4];
		t[1] = W[i + 8];
		t[2] = W[i + 24];
		t[3] = W[i + 60];
		W[i + 4] = W[i + 20];
		W[i + 8] = W[i + 40];
		W[i + 24] = W[i + 56];
		W[i + 60] = W[i + 44];

		W[i + 20] = W[i + 36];
		W[i + 40] = t[1];
		W[i + 56] = t[2];
		W[i + 44] = W[i + 28];

		W[i + 28] = W[i + 12];
		W[i + 12] = t[3];
		W[i + 36] = W[i + 52];
		W[i + 52] = t[0];
	}
	// Mix Columns
#pragma unroll 4
	for (int i = 0; i < 4; i++){ // Schleife über je 2*uint32_t
#pragma unroll 4
		for (int idx = 0; idx < 64; idx += 16){ // Schleife über die elemnte
			uint32_t a[4];
			a[0] = W[idx + i];
			a[1] = W[idx + i + 4];
			a[2] = W[idx + i + 8];
			a[3] = W[idx + i + 12];

			uint32_t ab = a[0] ^ a[1];
			uint32_t bc = a[1] ^ a[2];
			uint32_t cd = a[2] ^ a[3];

			uint32_t t, t2, t3;
			t = (ab & 0x80808080);
			t2 = (bc & 0x80808080);
			t3 = (cd & 0x80808080);

			uint32_t abx = (t >> 7) * 27U ^ ((ab^t) << 1);
			uint32_t bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			uint32_t cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[idx + i] = (bc^ a[3] ^ abx);
			W[idx + i + 4] = xor3(a[0], cd, bcx);
			W[idx + i + 8] = xor3(ab, a[3], cdx);
			W[idx + i + 12] = xor3(ab, a[2], xor3(abx, bcx, cdx));
		}
	}
}

__global__ __launch_bounds__(256, 3) /* will force 80 registers */
void x11_echo512_gpu_hash_64_final_sp(uint32_t threads, uint64_t *g_hash, uint32_t* resNonce, const uint64_t target)
{
	__shared__ __align__(16) uint32_t sharedMemory[8 * 1024];

	aes_gpu_init256_32(sharedMemory);


	const uint32_t P[48] = {
		0xe7e9f5f5, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af, 0xa4213d7e, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
		//8-12
		0x01425eb8, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af, 0x65978b09, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
		//21-25
		0x2cb6b661, 0x6b23b3b3, 0xcf93a7cf, 0x9d9d3751, 0x9ac2dea3, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
		//34-38
		0x579f9f33, 0xfbfbfbfb, 0xfbfbfbfb, 0xefefd3c7, 0xdbfde1dd, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
		0x34514d9e, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af, 0xb134347e, 0xea6f7e7e, 0xbd7731bd, 0x8a8a1968,
		0x14b8a457, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af, 0x265f4382, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af
		//58-61
	};
	uint32_t k0;
	uint32_t h[16];

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads){

		const uint32_t *hash = (uint32_t*)&g_hash[thread << 3];

		*(uint2x4*)&h[0] = __ldg4((uint2x4*)&hash[0]);
		*(uint2x4*)&h[8] = __ldg4((uint2x4*)&hash[8]);

		uint64_t backup = *(uint64_t*)&h[6];

		k0 = 512 + 8;

		__threadfence_block();

#pragma unroll 4
		for (uint32_t idx = 0; idx < 16; idx += 4)
			AES_2ROUND_32(sharedMemory, h[idx + 0], h[idx + 1], h[idx + 2], h[idx + 3], k0);

		k0 += 4;

		uint32_t W[64];

#pragma unroll 4
		for (uint32_t i = 0; i < 4; i++){
			uint32_t a = P[i];
			uint32_t b = P[i + 4];
			uint32_t c = h[i + 8];
			uint32_t d = P[i + 8];

			uint32_t ab = a ^ b;
			uint32_t bc = b ^ c;
			uint32_t cd = c ^ d;


			uint32_t t = ((a ^ b) & 0x80808080);
			uint32_t t2 = ((b ^ c) & 0x80808080);
			uint32_t t3 = ((c ^ d) & 0x80808080);

			uint32_t abx = ((t >> 7) * 27U) ^ ((ab^t) << 1);
			uint32_t bcx = ((t2 >> 7) * 27U) ^ ((bc^t2) << 1);
			uint32_t cdx = ((t3 >> 7) * 27U) ^ ((cd^t3) << 1);

			W[0 + i] = bc ^ d ^ abx;
			W[4 + i] = a ^ cd ^ bcx;
			W[8 + i] = ab ^ d ^ cdx;
			W[12 + i] = abx ^ bcx ^ cdx ^ ab ^ c;

			a = P[12 + i];
			b = h[i + 4];
			c = P[12 + i + 4];
			d = P[12 + i + 8];

			ab = a ^ b;
			bc = b ^ c;
			cd = c ^ d;


			t = (ab & 0x80808080);
			t2 = (bc & 0x80808080);
			t3 = (cd & 0x80808080);

			abx = (t >> 7) * 27U ^ ((ab^t) << 1);
			bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[16 + i] = abx ^ bc ^ d;
			W[16 + i + 4] = bcx ^ a ^ cd;
			W[16 + i + 8] = cdx ^ ab ^ d;
			W[16 + i + 12] = abx ^ bcx ^ cdx ^ ab ^ c;

			a = h[i];
			b = P[24 + i];
			c = P[24 + i + 4];
			d = P[24 + i + 8];

			ab = a ^ b;
			bc = b ^ c;
			cd = c ^ d;


			t = (ab & 0x80808080);
			t2 = (bc & 0x80808080);
			t3 = (cd & 0x80808080);

			abx = (t >> 7) * 27U ^ ((ab^t) << 1);
			bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[32 + i] = abx ^ bc ^ d;
			W[32 + i + 4] = bcx ^ a ^ cd;
			W[32 + i + 8] = cdx ^ ab ^ d;
			W[32 + i + 12] = abx ^ bcx ^ cdx ^ ab ^ c;

			a = P[36 + i];
			b = P[36 + i + 4];
			c = P[36 + i + 8];
			d = h[i + 12];

			ab = a ^ b;
			bc = b ^ c;
			cd = c ^ d;

			t = (ab & 0x80808080);
			t2 = (bc & 0x80808080);
			t3 = (cd & 0x80808080);

			abx = (t >> 7) * 27U ^ ((ab^t) << 1);
			bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[48 + i] = abx ^ bc ^ d;
			W[48 + i + 4] = xor3(bcx , a , cd);
			W[48 + i + 8] = xor3(cdx , ab, d);
			W[48 + i + 12] = xor3(abx , bcx , xor3(cdx, ab, c));


		}

		for (int k = 1; k < 10; k++)
			echo_round_sp(sharedMemory, W, k0);

#pragma unroll 4
		for (int i = 0; i < 16; i += 4)
		{
			W[i] ^= W[32 + i] ^ 512;
			W[i + 1] ^= W[32 + i + 1];
			W[i + 2] ^= W[32 + i + 2];
			W[i + 3] ^= W[32 + i + 3];
		}
		uint64_t check = ((uint64_t*)hash)[3] ^ ((uint64_t*)W)[3];

		if (check <= target)
		{
			uint32_t tmp = atomicExch(&resNonce[0], thread);
			if (tmp != UINT32_MAX)
				resNonce[1] = tmp;
		}
	}
}

__host__
void x11_echo512_cpu_hash_64_final_sp(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t *d_resNonce, const uint64_t target)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x11_echo512_gpu_hash_64_final_sp<<<grid, block>>>(threads, (uint64_t*)d_hash,d_resNonce,target);
}

__global__ __launch_bounds__(384, 2)
static void x11_echo512_gpu_hash_64_sp(uint32_t threads, uint32_t *g_hash)
{
	__shared__ uint32_t sharedMemory[8 * 1024];

	//	if (threadIdx.x < 256)
	//	{
	aes_gpu_init256_32(sharedMemory);
	//	}
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	uint32_t k0;
	uint32_t h[16];
	uint32_t hash[16];
	if (thread < threads){

		uint32_t *Hash = &g_hash[thread << 4];

		*(uint2x4*)&h[0] = __ldg4((uint2x4*)&Hash[0]);
		*(uint2x4*)&h[8] = __ldg4((uint2x4*)&Hash[8]);

		//		*(uint2x4*)&hash[0] = *(uint2x4*)&h[0];
		//		*(uint2x4*)&hash[8] = *(uint2x4*)&h[8];


		const uint32_t P[48] = {
			0xe7e9f5f5, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af, 0xa4213d7e, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
			//8-12
			0x01425eb8, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af, 0x65978b09, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
			//21-25
			0x2cb6b661, 0x6b23b3b3, 0xcf93a7cf, 0x9d9d3751, 0x9ac2dea3, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
			//34-38
			0x579f9f33, 0xfbfbfbfb, 0xfbfbfbfb, 0xefefd3c7, 0xdbfde1dd, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
			0x34514d9e, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af, 0xb134347e, 0xea6f7e7e, 0xbd7731bd, 0x8a8a1968,
			0x14b8a457, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af, 0x265f4382, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af
			//58-61
		};

		k0 = 520;
		__threadfence_block();

#pragma unroll 4
		for (uint32_t idx = 0; idx < 16; idx += 4)
		{
			AES_2ROUND_32(sharedMemory, h[idx + 0], h[idx + 1], h[idx + 2], h[idx + 3], k0);
		}
		k0 += 4;

		uint32_t W[64];

#pragma unroll 4
		for (uint32_t i = 0; i < 4; i++)
		{
			uint32_t a = P[i];
			uint32_t b = P[i + 4];
			uint32_t c = h[i + 8];
			uint32_t d = P[i + 8];

			uint32_t ab = a ^ b;
			uint32_t bc = b ^ c;
			uint32_t cd = c ^ d;


			uint32_t t = (ab & 0x80808080);
			uint32_t t2 = (bc & 0x80808080);
			uint32_t t3 = (cd & 0x80808080);

			uint32_t abx = (t >> 7) * 27U ^ ((ab^t) << 1);
			uint32_t bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			uint32_t cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[i] = abx ^ bc ^ d;
			W[i + 4] = bcx ^ a ^ cd;
			W[i + 8] = cdx ^ ab ^ d;
			W[i + 12] = abx ^ bcx ^ cdx ^ ab ^ c;

			a = P[i + 12];
			b = h[i + 4];
			c = P[i + 16];
			d = P[i + 20];

			ab = a ^ b;
			bc = b ^ c;
			cd = c ^ d;


			t = (ab & 0x80808080);
			t2 = (bc & 0x80808080);
			t3 = (cd & 0x80808080);

			abx = (t >> 7) * 27U ^ ((ab^t) << 1);
			bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[16 + i] = bc ^ d ^ abx;
			W[16 + i + 4] = a ^ cd ^ bcx;
			W[16 + i + 8] = d ^ ab ^ cdx;
			W[16 + i + 12] = c ^ ab ^ abx ^ bcx ^ cdx;

			a = h[i];
			b = P[24 + i + 0];
			c = P[24 + i + 4];
			d = P[24 + i + 8];

			ab = a ^ b;
			bc = b ^ c;
			cd = c ^ d;


			t = (ab & 0x80808080);
			t2 = (bc & 0x80808080);
			t3 = (cd & 0x80808080);

			abx = (t >> 7) * 27U ^ ((ab^t) << 1);
			bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[32 + i] = bc ^ d ^ abx;
			W[32 + i + 4] = a ^ cd ^ bcx;
			W[32 + i + 8] = d ^ ab ^ cdx;
			W[32 + i + 12] = c ^ ab ^ abx ^ bcx ^ cdx;

			a = P[36 + i];
			b = P[36 + i + 4];
			c = P[36 + i + 8];
			d = h[i + 12];

			ab = a ^ b;
			bc = b ^ c;
			cd = c ^ d;

			t = (ab & 0x80808080);
			t2 = (bc & 0x80808080);
			t3 = (cd & 0x80808080);

			abx = (t >> 7) * 27U ^ ((ab^t) << 1);
			bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[48 + i] = (bc ^ d ^ abx);
			W[48 + i + 4] = (a ^ cd ^ bcx);
			W[48 + i + 8] = (d^ ab^ cdx);
			W[48 + i + 12] = (c ^ ab ^ (abx ^ bcx ^cdx));

		}

		for (int k = 1; k < 10; k++)
			echo_round_sp(sharedMemory, W, k0);

#pragma unroll 4
		for (int i = 0; i < 16; i += 4)
		{
			W[i] ^= W[32 + i] ^ 512;
			W[i + 1] ^= W[32 + i + 1];
			W[i + 2] ^= W[32 + i + 2];
			W[i + 3] ^= W[32 + i + 3];
		}
		*(uint2x4*)&Hash[0] = *(uint2x4*)&Hash[0] ^ *(uint2x4*)&W[0];
		*(uint2x4*)&Hash[8] = *(uint2x4*)&Hash[8] ^ *(uint2x4*)&W[8];
	}
}

__host__
void x11_echo512_cpu_hash_64_sp(int thr_id, uint32_t threads, uint32_t *d_hash){

	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x11_echo512_gpu_hash_64_sp<<<grid, block>>>(threads, d_hash);
}
